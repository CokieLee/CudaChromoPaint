#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void ColumnUpdateKernel(int* colReads, int numHaps,
    double probTransition, double probError, int* prevStateProbs, int* currStateProbs)
{
    int i = threadIdx.x;
    for (int hap = 0; hap < numHaps; hap++) {
        if (hap = i) {
            currStateProbs[i] = (1 - probTransition) * probError;
        }
        else {
            currStateProbs[i] = probTransition * probError;
        }
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t KernelTime(int* colReads, int numHaps,
    double probTransition, double probError, int* prevStateProbs, int* currStateProbs, double* timeSec)
{
    int* dev_reads = 0;
    int* dev_prevStateProbs = 0;
    int* dev_currStateProbs = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_reads, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_prevStateProbs, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_currStateProbs, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_reads, colReads, numHaps * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_prevStateProbs, prevStateProbs, numHaps * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    ColumnUpdateKernel << <1, numHaps >> > (dev_reads, numHaps, probTransition, probError, dev_prevStateProbs, dev_currStateProbs);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(currStateProbs, dev_currStateProbs, numHaps * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_reads);
    hipFree(dev_prevStateProbs);
    hipFree(dev_currStateProbs);

    return cudaStatus;
}

int main()
{
    const int minNumPeep = 10;
    const int maxNumPeep = 10'000;
    const int numTests = 300;

    const int seed = 4;

    const string outFile = "time.log";

    const int colSize = 5;
    const int a[colSize];
    for () {

    }

}
