#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__global__ void ColumnUpdateKernel(int* colReads, int numHaps,
    double probTransition, double probError, float* prevStateProbs, float* currStateProbs)
{
    int i = threadIdx.x;
    for (int hap = 0; hap < numHaps; hap++) {
        if (hap == i) {
            currStateProbs[i] = (1 - probTransition) * probError;
        }
        else {
            currStateProbs[i] = probTransition * probError;
        }
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t KernelTime(int* colReads, int numHaps,
    double probTransition, double probError, float* prevStateProbs, float* currStateProbs, double* timeSec)
{
    int* dev_reads = 0;
    float* dev_prevStateProbs = 0;
    float* dev_currStateProbs = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_reads, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_prevStateProbs, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_currStateProbs, numHaps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_reads, colReads, numHaps * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_prevStateProbs, prevStateProbs, numHaps * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    clock_t before = clock();
    ColumnUpdateKernel<<< 1, numHaps >>>(dev_reads, numHaps, probTransition, probError, dev_prevStateProbs, dev_currStateProbs);
    hipDeviceSynchronize();
    clock_t difference = clock() - before;
    *timeSec = difference * 1000 / CLOCKS_PER_SEC;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(currStateProbs, dev_currStateProbs, numHaps * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_reads);
    hipFree(dev_prevStateProbs);
    hipFree(dev_currStateProbs);

    return cudaStatus;
}

int main()
{
    const int minNumPeep = 10;
    const int maxNumPeep = 10'000;
    const int numTests = 300;

    const int seed = 4;
    srand(seed);

    char outFile[] = "time.log";

    const int numHaps = 10;
    const float probTransition = 0.05;
    const float probError = 0.005;

    double timeSec;

    float prevStateProbs[numHaps] = {};
    float currStateProbs[numHaps] = {};
    int colReads[numHaps] = {};

    for (int i = 0; i < numHaps; i++) {
        colReads[i] = rand() % 2;
    }

    printf("print \n");

    KernelTime(colReads, numHaps, probTransition, probError, prevStateProbs, currStateProbs, &timeSec);

    printf("time: %f\n", timeSec);
}
